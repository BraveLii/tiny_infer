#include <hip/hip_runtime.h>

#include <iostream>


__global__ void nchw_to_nhwc(float* dst, float* src, int num, int N, int C, int H, int W) {
    int num_td = gridDim.x * blockDim.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=idx; idx<num; idx+=num_td) {
        int n = i/(C*H*W);
        int c = i/(H*W)%C;
        int h = i%(H*W)/W;
        int w = i%W;

        dst[n*H*W*C + h*W*C + w*C + c] = src[i];
    }

    return;
}

__global__ void nchw_to_nhwc_naive(float* dst, float* src, int num, int N, int C, int H, int W) {
    int n = blockIdx.x;
    int c = blockIdx.y;
    int h = threadIdx.x;
    int w = threadIdx.y;

    int src_idx = n*H*W*C + c*H*W + h*W + w;
    if(src_idx < num) {
        dst[n*H*W*C + h*W*C + w*C + c] = src[src_idx];
    }

    return;
}

void print_matrix(float* data, int N, int C, int H, int W) {
    int len = N*C*H*W;
    for(int i=0; i<len; i++) {
        std::cout << data[i] << " ";

        // std::cout << "aa : " << (i+1)  << std::endl;
        if((i+1) % W == 0) {
            std::cout << std::endl;
        }

        if((i+1) % (H*W) == 0) {
            std::cout << std::endl;
        }

        if((i+1) % (C*H*W) == 0) {
            std::cout << "-------------" << std::endl;
        }
    }

    return;
}

int main() {
    int N = 1;
    int C = 2;
    int H = 3;
    int W = 3;

    int len = N*C*H*W;
    float *src = new float[len];

    for(int i=0; i<len; i++) {
        src[i] = i+1;
    }

    for(int i=0; i<len; i++) {
        std::cout << src[i] << " ";
    }
    std::cout << std::endl;

    float *dst = new float[len];

    float *d_src = nullptr;
    float *d_dst = nullptr;

    hipMalloc((void**)&d_src, len*sizeof(float));
    hipMalloc((void**)&d_dst, len*sizeof(float));
    hipMemcpy(d_src, src, len*sizeof(float), hipMemcpyHostToDevice);
    // int td_num = 512;
    // int block_num = (len+512-1)/512;

    // nchw_to_nhwc<<<block_num, td_num>>>(d_dst, d_src, len, N, C, H, W);

    dim3 grid(N,C);
    dim3 block(H,W);
    nchw_to_nhwc_naive<<<grid, block>>>(d_dst, d_src, len, N, C, H, W);

    hipMemcpy(dst, d_dst, len*sizeof(float), hipMemcpyDeviceToHost);
    

    for(int i=0; i<len; i++) {
        std::cout << dst[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "src ====>" << std::endl;
    print_matrix(src, N, C, H, W);
    std::cout << "dst ====>" << std::endl;
    print_matrix(dst, N, H, W, C);

    return 0;
}